#include "hip/hip_runtime.h"
#include "EpiG.h"

/****************************************
* Generates a circuit and executes |phi> = U|000> multiplication. Then it evaluates |phi> using 
* the formula from Coffman  that quantifies the amount of entanglement. 
 ****************************************/
void* GA::doEntanglState(qGate *myfinalG, hipComplex *entang, bool display) {

	float val = (float) finalGate->valuedness;
	float numIO = (float) finalGate->numIO;
	int l = (int(pow(val, numIO)));
	int x = (int(pow(val, numIO)));
	int y = (int(pow(val, numIO)));
	hipComplex logicresult[(int(pow(val, numIO)))];
	short h[2][2][2];

	//a sort of hash code
	h[0][0][0] = 0;
	h[0][0][1] = 1;
	h[0][1][0] = 2;
	h[0][1][1] = 3;
	h[1][0][0] = 4;
	h[1][0][1] = 5;
	h[1][1][0] = 6;
	h[1][1][1] = 7;

	long time = clock();
	//diplay code
	(hipMemcpy(d_MV, myfinalG->gateMatrix1, l*l*sizeof(hipComplex), hipMemcpyHostToDevice));
	if (display){
		for (int a = 0; a < l; a++){
			for (int b = 0; b < l; b++){
				cout << "(" << hipCrealf(myfinalG->gateMatrix1[a*l+b])<< "," << hipCimagf(myfinalG->gateMatrix1[a*l+b]) << ")"<< "";
			}
			cout<<endl;
		}
	}

	//execute U|00...0>
	vectorInitInit(l, d_VI);
	Mulvnoset(l, d_MV, d_VI, d_VO);
	hipMemcpy(logicresult, d_VO, l*sizeof(hipComplex), hipMemcpyDeviceToHost);

	if (display){
		cout << "O: ";
		for (int w = 0; w < l; w++)
			cout<<"("<<hipCrealf(logicresult[w])<<","<<hipCimagf(logicresult[w])<<")";
		cout<<endl;
	}

	hipComplex a000 = logicresult[0];
	hipComplex a001 = logicresult[1];
	hipComplex a010 = logicresult[2];
	hipComplex a011 = logicresult[3];
	hipComplex a100 = logicresult[4];
	hipComplex a101 = logicresult[5];
	hipComplex a110 = logicresult[6];
	hipComplex a111 = logicresult[7];

	entang[0] = hipCaddf(hipCaddf(hipCaddf(hipCmulf(hipCmulf(a000,a000),hipCmulf(a111,a111)),hipCmulf(hipCmulf(a001,a001),hipCmulf(a110,a110))),hipCmulf(hipCmulf(a010,a010),hipCmulf(a101,a101))),hipCmulf(hipCmulf(a100,a100),hipCmulf(a011,a011)));
	entang[1] = hipCaddf(hipCaddf(hipCaddf(hipCaddf(hipCaddf(hipCmulf(hipCmulf(hipCmulf(a000,a111),a011),a100),hipCmulf(hipCmulf(hipCmulf(a000,a111),a101),a010)),hipCmulf(hipCmulf(hipCmulf(a000,a111),a110),a001)),hipCmulf(hipCmulf(hipCmulf(a011,a100),a101),a010)),hipCmulf(hipCmulf(hipCmulf(a011,a100),a110),a001)),hipCmulf(hipCmulf(hipCmulf(a101,a010),a110),a001));
	entang[2] = hipCaddf(hipCmulf(hipCmulf(hipCmulf(a000,a110),a101),a011),hipCmulf(hipCmulf(hipCmulf(a111,a001),a010),a100));


	hipComplex I5 = {0,0};


	for (int i1 = 0;i1<2;i1++)
	for (int j1 = 0;j1<2;j1++)
	for (int k1 = 0;k1<2;k1++)
	for (int i2 = 0;i2<2;i2++)
	for (int j2 = 0;j2<2;j2++)
	for (int k2 = 0;k2<2;k2++)
	for (int i3 = 0;i3<2;i3++)
	for (int j3 = 0;j3<2;j3++)
	for (int k3 = 0;k3<2;k3++){
		I5 = hipCaddf(I5,hipCmulf(logicresult[h[i1][j1][k1]],hipCmulf(logicresult[h[i2][j2][k2]],hipCmulf(logicresult[h[i3][j3][k3]],hipCmulf(hipConjf(logicresult[h[i1][j2][k3]]),hipCmulf(hipConjf(logicresult[h[i2][j3][k1]]), hipConjf(logicresult[h[i3][j1][k2]])))))));	
	}
	entang[6] = I5;

	hipComplex I2 = {0,0};
	hipComplex I3 = {0,0};
	hipComplex I4 = {0,0};

	for (int i1 = 0;i1<2;i1++)
	for (int j1 = 0;j1<2;j1++)
	for (int k1 = 0;k1<2;k1++)
	for (int i2 = 0;i2<2;i2++)
	for (int j2 = 0;j2<2;j2++)
	for (int k2 = 0;k2<2;k2++){
		I2 = hipCaddf(I2,hipCmulf(logicresult[h[i1][j1][k1]],hipCmulf(logicresult[h[i2][j2][k2]],hipCmulf(hipConjf(logicresult[h[i1][j1][k2]]),hipConjf(logicresult[h[i2][j2][k1]])))));
		I3 = hipCaddf(I3,hipCmulf(logicresult[h[i1][j1][k1]],hipCmulf(logicresult[h[i2][j2][k2]],hipCmulf(hipConjf(logicresult[h[i1][j2][k1]]),hipConjf(logicresult[h[i2][j1][k2]])))));
		I4 = hipCaddf(I4,hipCmulf(logicresult[h[i1][j1][k1]],hipCmulf(logicresult[h[i2][j2][k2]],hipCmulf(hipConjf(logicresult[h[i1][j2][k2]]),hipConjf(logicresult[h[i2][j1][k1]])))));
	}
	
	entang[3] = I2;
	entang[4] = I3;
	entang[5] = I4;
	if (display){
		cout << "D4: ";
//			cout<<"("<<hipCrealf(d4)<<","<<hipCimagf(d4)<<")";
		cout<<endl;
	}
	
	return 0;
}









/****************************************
* Generates a set of states representing a given sequence: the input qubit is the one before the last from the bottom
* the output qubit is the bottom one
* both input and output qubits must be measured after each operation in order to be able to initialize them properly
* for the next step.
* A sequence is detected when for all input elements the output is |0> and for the last input value the output is |1>
 ****************************************/
void* GA::doMeasureFASeqFitness(Individual *ind, bool display) {

	//define temporal variables for the results and the manupaltion
	//of the ga_measurement
	qGate *myfinalG;
	float val = (float) finalGate->valuedness;
	float numIO = (float) finalGate->numIO;
	float err,maxcount = 0;
	int l = (int(pow(val, numIO)));
	int x = (int(pow(val, numIO)));
	int y = (int(pow(val, numIO)));
	hipComplex inter[(int(pow(val, numIO)))];
	hipComplex inter_b[(int(pow(val, numIO)))];
	hipComplex logicresult[(int(pow(val, numIO)))];
	hipComplex stb[(int(pow(val, numIO)))];
	hipComplex expectations[(int(pow(val, numIO) * val))], cu_alphas;
	hipComplex resultingTotalError[(int(pow(val, numIO)))][ga_measurement * (int) val];
	hipComplex resultingTotal[(int(pow(val, numIO)))][ga_measurement * (int) val];
	complex<float> temp_val;

	Individual *indi = ind;
	indi->Error = 0;
	indi->fitness = 0;
	indi->Cost = 0;

	//threaded code
	long time = clock();
	//accelerated matrix computation
	myfinalG = computeCUDAMatrix(indi, false);
	(hipMemcpy(d_MV, myfinalG->gateMatrix1, l*l*sizeof(hipComplex), hipMemcpyHostToDevice));
	numIO = (float) myfinalG->numIO;
	//check ifga_phase is defined and multiply the whole matrix by it
	if (ga_phase == 1) {
		maxcount = (int(pow(pow(val, numIO),2)));
		if (hipCrealf(indi->phase) != 0 && hipCimagf(indi->phase) != 0) {
			for (int k = 0; k < maxcount; k++)
					myfinalG->gateMatrix1[k] = hipCmulf(
							myfinalG->gateMatrix1[k], indi->phase);
		}
	}

if (display){
		out_stream<<indi->my_string<<endl;
}

	//for all values in the desireed sequence


	(hipMemcpy(myfinalG->gateMatrix1, d_MV, l*l*sizeof(hipComplex), hipMemcpyDeviceToHost));

	vectorInitZero(l, d_VI);

	for (int k = 0; k < measured_desired_output_records; k++) {


		//empty all intermediary variables
		for (int w = 0; w < l; w++)
			inter[w] = logicresult[w] = stb[w] = make_hipFloatComplex(0, 0);

		vectorSetSeqNext(k, l, sequence_desired[k][0], d_VI);
if (display){

		hipMemcpy(stb, d_VI, l*sizeof(hipComplex), hipMemcpyDeviceToHost);
		cout << "I: ";
		for (int w = 0; w < l; w++)
			cout<<"("<<hipCrealf(stb[w])<<","<<hipCimagf(stb[w])<<")";
		cout<<endl;
}
		
		Mulvnoset(l, d_MV, d_VI, d_VO);


if (display){
		cout << "O: ";
		hipMemcpy(logicresult, d_VO, l*sizeof(hipComplex), hipMemcpyDeviceToHost);
		for (int w = 0; w < l; w++)
			cout<<"("<<hipCrealf(logicresult[w])<<","<<hipCimagf(logicresult[w])<<")";
		cout<<endl;
}
		//measure for only the desired state of the single output qubit
		for (int w = 0; w < l; w++)
			inter[w] = inter_b[w] = make_hipFloatComplex(0, 0);


		//measure for 0 when non final sequence element
		if (k != measured_desired_output_records-1){
			Mulvnoset(l, d_ME_00, d_VO, d_VI);
		}else{
			//measure for 1 when the end of the sequence was reached
			Mulvnoset(l, d_ME_01, d_VO, d_VI);
		}


if (display){
		cout << "Measure Output: ";
		hipMemcpy(inter, d_VI, l*sizeof(hipComplex), hipMemcpyDeviceToHost);
		for (int w = 0; w < l; w++)
			cout<<"("<<hipCrealf(inter[w])<<","<<hipCimagf(inter[w])<<")";
		cout<<endl;
}

		expectations[0] = make_hipFloatComplex(0, 0);
		//the finnal inner product for p(r) - generate the expectation for the measured output
		Mulconjvvnoset(l,d_VI, d_VI, d_Value);

		//set the probability of obtaining the desired output for this input value
		hipMemcpy(&resultingTotal[k][0], &d_Value[0], sizeof(hipComplex), hipMemcpyDeviceToHost);
				
		//skip don't cares
		if (hipCrealf(sequence_desired[k][1]) == 0 && hipCimagf(sequence_desired[k][1]) == -1) {
			resultingTotalError[k][(int) val]     = make_hipFloatComplex(0, 0);
		} else {
			//calculate the error if this is not to be considered as don't care
			resultingTotalError[k][(int) val] = make_hipFloatComplex(sqrt(pow(hipCrealf(hipCsubf(resultingTotal[k][0],sequence_desired[k][1])),(float) 2)), 0);
		}


		temp_val = complex<float> (hipCrealf(expectations[0]), hipCimagf(expectations[0]));
		temp_val = sqrt(temp_val);
		cu_alphas = make_hipFloatComplex(real(temp_val), imag(temp_val));

		//if the ga_measurement generated at least some output
		if (!(hipCrealf( cu_alphas) == 0 && hipCimagf( cu_alphas) == 0)){
			//if the output qubit was measured in 0, normalize the vector
			hipMemcpy(&d_Value[0], &cu_alphas, sizeof(hipComplex),hipMemcpyHostToDevice);
			Divvnoset(l, d_VI, d_Value, d_VO);
			hipMemcpy(d_VI, d_VO, sizeof(hipComplex),hipMemcpyDeviceToDevice);
		} else {
			//if the ga_measurement was not succesful, use the output of the circuit 
		}



		//measure the input qubit for 0 - to reset it and allow consequent initialization
		if (k < measured_desired_output_records - 1)
			if (hipCrealf(sequence_desired[k+1][0]) == 0){
				Mulvnoset(l, d_ME_10, d_VI, d_VO);
			} else {
				Mulvnoset(l, d_ME_11, d_VI, d_VO);
			}	

if (display){
		hipMemcpy(inter_b, d_VO, l*sizeof(hipComplex), hipMemcpyDeviceToHost);
		cout << "Measure Input: ";
		for (int w = 0; w < l; w++)
			cout<<"("<<hipCrealf(inter_b[w])<<","<<hipCimagf(inter_b[w])<<")";
		cout<<endl;
}
		expectations[0] = make_hipFloatComplex(0, 0);
		//calculate the expectation value for measureing the qubit in state |0>
		Mulconjvvnoset(l,d_VO, d_VO, d_Value);
		hipMemcpy(expectations, d_Value, sizeof(hipComplex), hipMemcpyDeviceToHost);
		temp_val = complex<float> (hipCrealf(expectations[0]), hipCimagf(expectations[0]));
		temp_val = sqrt(temp_val);
		cu_alphas = make_hipFloatComplex(real(temp_val), imag(temp_val));

		hipMemcpy(d_VI, d_VO, l*sizeof(hipComplex), hipMemcpyDeviceToDevice);

		//if the input qubit was measured in 0, normalize the vector
		if (!(hipCrealf( cu_alphas) == 0 && hipCimagf( cu_alphas) == 0)){
			hipMemcpy(&d_Value[0], &cu_alphas, sizeof(hipComplex),hipMemcpyHostToDevice);
			Divvnoset(l, d_VI, d_Value, d_VI);

		}

	}

	int m;
	err = 0;
	//calculate the overall error
	for (int c = 0; c < ga_measurement; c++) {
		m = measured_desired_output_records;
		inter[0] = make_hipFloatComplex(0, 0);
		for (int e = 0; e < measured_desired_output_records; e++) {
			//inter[0] = hipCmulf(inter[0], resultingTotal[e][0]);
			inter[0] = hipCaddf(inter[0], make_hipFloatComplex(hipCrealf(resultingTotalError[e][(int) val]), 0));
		}
		err += hipCrealf(hipCdivf(inter[0], make_hipFloatComplex(m, 0)));
	}
	err /= ga_measurement;
	indi->Error = err;

	//generate fitness value
	if (indi->Error != 0) {
		switch (ga_replicator) {
		case 0:
			//simple fitness1
			indi->fitness = (1 - indi->Error);
			break;
		case 1:
			//simple fitness1
			indi->fitness = (1 / (indi->Error + 1));
			break;
		case 2:
			//scaled complex fitness1
			indi->fitness = (ga_alpha * (1 - indi->Error) + ga_beta * (exp(-pow(abs(
					ga_divider - indi->Cost), 2))));
			break;
		case 3:
			//scaled complex fitness2
			indi->fitness = (ga_alpha * (1 / (indi->Error + 1)) + ga_beta * (exp(
					-pow(abs(ga_divider - indi->Cost), 2))));
			break;
		case 4:
			indi->fitness = (1 - indi->Error);
			break;
		case 5:
			indi->fitness = exp(-indi->Error);
			break;
		}
	} else {
		indi->Error = 0;
		indi->fitness = 1;//we found a good individual;
	}
	if (display) {

		out_stream<< "Circuit Matrix:"<<endl;
		for (int i = 0; i < l; i++) {
			for (int j = 0; j < l; j++) {
				out_stream<<"("<<hipCrealf(myfinalG->gateMatrix1[i*l+j])<<","<<hipCimagf(myfinalG->gateMatrix1[i*l+j])<<")";
			}
			out_stream<<endl;
		}
		out_stream<<endl;


		out_stream << "Error: " << indi->Error << endl;
		out_stream << "Fitness: " << indi->fitness << endl;
		out_stream << "Valuedness: " << indi->valuedness << endl;
		out_stream << endl;
		out_stream << "Sequence length: "<<measured_desired_output_records<<endl;;
		out_stream << "Sequence: ";

		for (int m = 0; m < measured_desired_output_records; m++) 
			out_stream<<"  "<<hipCrealf(sequence_desired[m][0])<<"  ";
		
		out_stream << endl<<" Desired: ";
		for (int m = 0; m < measured_desired_output_records; m++) 
			out_stream<<"("<<hipCrealf(sequence_desired[m][1])<<","<<hipCimagf(sequence_desired[m][1])<<")";


		out_stream << endl<<"Observables: ";
		for (int m = 0; m < measured_desired_output_records-1; m++) 
			out_stream<<"  0  ";
		out_stream<<"  1  ";

		out_stream << endl<<"Obtained: ";
		for (int m = 0; m < measured_desired_output_records; m++) 
			out_stream<<"("<<hipCrealf(resultingTotal[m][0])<<","<<hipCimagf(resultingTotal[m][0])<<")";

		out_stream << endl <<" -------- " << endl;
	}

	time = clock() -time ;
	//cout<<"CUBLAS fitness done: "<<(float)(time/CLOCKS_PER_SEC)<<", for "<<indi->ioNumber<<" qubits "<<endl;
	destroyGate(myfinalG);
	delete (myfinalG);
	return 0;
}

/****************************************
 * calculates theprobabilities for obtaining |0> and |1>
 * Right now it is doing ga_measurement in increasing order
 * of all defined bits to measure
 ****************************************/
void* GA::doMeasureFitness(Individual *ind, bool display) {

	//define temporal variables for the results and the manupaltion
	//of the ga_measurement
	qGate *measures[10], *myfinalG;
	float val = (float) finalGate->valuedness;
	float numIO = (float) finalGate->numIO;
	float err,maxcount = 0;
	int l = (int(pow(val, numIO)));
	int x = (int(pow(val, numIO)));
	int y = (int(pow(val, numIO)));
	hipComplex inter[(int(pow(val, numIO)))];
	hipComplex logicresult[(int(pow(val, numIO)))];
	hipComplex sta[(int(pow(val, numIO)))];
	hipComplex expectations[(int(pow(val, numIO) * val))];
	hipComplex resultingTotalError[(int(pow(val, numIO)))][ga_measurement
	                                                      * (int) val];
	hipComplex resultingTotal[(int(pow(val, numIO)))][ga_measurement * (int) val];

	Individual *indi = ind;
	indi->Error = 0;
	indi->fitness = 0;
	indi->Cost = 0;

	//myfinalG = computeMatrix(indi);
	//cout << indi->my_string<<endl;
	long time = clock();
	myfinalG = computeCUDAMatrix(indi,false);
	numIO = (float) myfinalG->numIO;
	//check ifga_phase is defined and multiply the whole matrix by it
	if (ga_phase == 1) {
		maxcount = (int(pow(pow(val, numIO),2)));
		if (hipCrealf(indi->phase) != 0 && hipCimagf(indi->phase) != 0) {
			for (int k = 0; k < maxcount; k++)
					myfinalG->gateMatrix1[k] = hipCmulf(
							myfinalG->gateMatrix1[k], indi->phase);
		}
	}


	//for single qubit this should always be only 0, i.e. <1
	for (int m = 0; m < ga_measurement; m++) {
		for (int r = 0; r < val; r++) {
			measures[r] = measurements[(measurementQBits[m] * (int) val) + r];
		}
		//for all input states l
		for (int k = 0; k < measured_desired_output_records; k++) {

			for (int w = 0; w < l; w++)
				sta[w] = inter[w] = logicresult[w] = make_hipFloatComplex(0, 0);
			//set the k-th input state - minterm
			sta[measured_desired_output_records_idx[k]] = make_hipFloatComplex(1, 0);

			//propagate the state through the matrix
			for (int i = 0; i < l; i++) {
				for (int j = 0; j < l; j++) {
					logicresult[i] = hipCaddf(logicresult[i], hipCmulf(sta[j],
							myfinalG->gateMatrix1[i*l+j]));
				}
			}
			//measure for each state of the single given qubit
			for (int r = 0; r < val; r++) {
				for (int w = 0; w < l; w++)
					inter[w] = make_hipFloatComplex(0, 0);
				for (int i = 0; i < l; i++) {
					//rows
					for (int j = 0; j < l; j++) {
						inter[i] = hipCaddf(inter[i], hipCmulf(logicresult[j],
								measures[r]->gateMatrix1[i*l+j]));
					}
				}


				//the finnal inner product for p(r)
				expectations[((m * (int) val) + r)] = make_hipFloatComplex(0, 0);
				for (int j = 0; j < l; j++) {
					expectations[(m * (int) val + r)] = hipCaddf(expectations[(m
							* (int) val + r)], hipCmulf(hipConjf(logicresult[j]),
									inter[j]));
				}

				resultingTotal[measured_desired_output_records_idx[k]][(m * (int) val + r)] = expectations[(m
						* (int) val + r)];
				
				if (hipCrealf(measureexpected[measured_desired_output_records_idx[k]][(int) val * m + r]) == 0
						&& hipCimagf(measureexpected[measured_desired_output_records_idx[k]][(int) val * m + r]) == 1) {
					//skip don't cares
					resultingTotalError[measured_desired_output_records_idx[k]][m * (int) val + r]
					                       = make_hipFloatComplex(0, 0);
				} else {
					resultingTotalError[measured_desired_output_records_idx[k]][m * (int) val + r]
					                       = make_hipFloatComplex(sqrt(pow(hipCrealf(hipCsubf(
					                    		   resultingTotal[measured_desired_output_records_idx[k]][(m * (int) val) + r],
					                    		   measureexpected[measured_desired_output_records_idx[k]][((int) val * m) + r])),
					                    		   (float) 2)), 0);
				}
				//for (int i = 0; i < l; i++) 

			}
		}
	}

	int m;
	err = 0;
	for (int c = 0; c < ga_measurement; c++) {
		m = l * (int) val;
		inter[0] = make_hipFloatComplex(0, 0);
		for (int e = 0; e < measured_desired_output_records; e++) {
			for (int r = 0; r < val; r++) {
				if (!(hipCrealf(measureexpected[measured_desired_output_records_idx[e]][(int) val * c + r]) == 0
						&& hipCimagf(measureexpected[measured_desired_output_records_idx[e]][(int) val * c + r])	== -1))
					inter[0] = hipCaddf(inter[0], make_hipFloatComplex(hipCrealf(
							resultingTotalError[measured_desired_output_records_idx[e]][c * (int) val + r]), 0));
				else
					m -= (int) val;
			}
		}
		err += hipCrealf(hipCdivf(inter[0], make_hipFloatComplex(m, 0)));
	}
	err /= ga_measurement;
	indi->Error = err;

	//generate fitness value
	if (indi->Error != 0) {
		switch (ga_replicator) {
		case 0:
			//simple fitness1
			indi->fitness = (1 - indi->Error);
			break;
		case 1:
			//simple fitness1
			indi->fitness = (1 / (indi->Error + 1));
			break;
		case 2:
			//scaled complex fitness1
			indi->fitness = (ga_alpha * (1 - indi->Error) + ga_beta * (exp(-pow(abs(
					ga_divider - indi->Cost), 2))));
			break;
		case 3:
			//scaled complex fitness2
			indi->fitness = (ga_alpha * (1 / (indi->Error + 1)) + ga_beta * (exp(
					-pow(abs(ga_divider - indi->Cost), 2))));
			break;
		case 4:
			indi->fitness = (1 - indi->Error);
			break;
		case 5:
			indi->fitness = exp(-indi->Error);
			break;
		}
	} else {
		indi->Error = 0;
		indi->fitness = 1;//we found a good individual;
	}
	if (display) {
		out_stream << "Error: " << indi->Error << endl;
		out_stream << "Fitness: " << indi->fitness << endl;
		out_stream << "Valuedness: " << indi->valuedness << endl;
		out_stream << endl;
		out_stream << "Input St. Des_0 Des_1 Out_0 Out_1 Error_0 Error_1 "
		<< endl;

		for (int m = 0; m < ga_measurement; m++) {
			//for all input states l
			for (int k = 0; k < measured_desired_output_records; k++) {
				out_stream << "  " << measured_desired_output_records_idx[k] << "   ";
				for (int r = 0; r < val; r++) {
					out_stream << "  (" << hipCrealf(
							measureexpected[measured_desired_output_records_idx[k]][(int) val * m + r]) << ","
							<< hipCimagf(measureexpected[measured_desired_output_records_idx[k]][(int) val * m + r])
							<< ")";
				}
				//measure for each state of the single given qubit
				for (int r = 0; r < val; r++) {
					out_stream << "  (" << hipCrealf(resultingTotal[measured_desired_output_records_idx[k]][(m
							* (int) val + r)]) << "," << hipCimagf(
									resultingTotal[measured_desired_output_records_idx[k]][(m * (int) val + r)]) << ")";
				}
				for (int r = 0; r < val; r++) {
					out_stream << "  (" << hipCrealf(resultingTotalError[measured_desired_output_records_idx[k]][m
					                                                       * (int) val + r]) << "," << hipCimagf(
					                                                    		   resultingTotalError[measured_desired_output_records_idx[k]][m * (int) val + r]) << ")";
				}
				out_stream << endl;
			}
		}
		out_stream << " -------- " << endl;
	}

	time = clock() -time ;
//	cout<<"CUBLAS fitness done: "<<(float)(time/CLOCKS_PER_SEC)<<", for "<<indi->ioNumber<<" qubits "<<endl;
	destroyGate(myfinalG);
	delete (myfinalG);
	return 0;
}

/****************************************
 * calculates the probabilities for obtaining two distinct multi qubit states of the form
 * (p_0 + p_1 + .. + p_k)|x_0...x_k> and (r_0 + r_1 + .. + r_k)|y_0....y_k>
 ****************************************/
void* GA::doMultiMeasureFitness(Individual *ind, bool display) {

	//define temporal variables for the results and the manipulation
	//of the ga_measurement
	int l = (int(pow((float) 2, (float) finalGate->numIO)));
	int x = (int(pow((float) 2, (float) finalGate->numIO)));
	int y = (int(pow((float) 2, (float) finalGate->numIO)));
	int maxcount = 0;
	int mes = 0;
	hipComplex inter0[(int(pow((float) 2, (float) finalGate->numIO)))];
	hipComplex inter1[(int(pow((float) 2, (float) finalGate->numIO)))];
	hipComplex logicresult[(int(pow((float) 2, (float) finalGate->numIO)))];
	hipComplex sta[(int(pow((float) 2, (float) finalGate->numIO)))];
	hipComplex expectation0, expectation1, ga_alphas, ga_betas;
	complex<float> expe_0, expe_1;
	hipComplex resultingTotalError[(int(pow((float) 2, (float) finalGate->numIO)))][ga_measurement* 2];
	hipComplex resultingTotal[(int(pow((float) 2, (float) finalGate->numIO)))][ga_measurement* 2];

	hipComplex inter, expect0, expect1;
//	qGate *measure0, *measure1;
	qGate *myfinalG, measure;

	Individual *indi = ind;
	//null this Individual previous evaluation
	indi->Error = 0;
	indi->fitness = 0;
	indi->Cost = 0;

	//threaded code
	int rc = pthread_mutex_lock(&mtx);
	if (rc) {
		cout << "Matrix locked: " << ind << endl;
	}
	// compute hte matrix of the circuit
	//myfinalG = computeMatrix(indi);
	myfinalG = computeCUDAMatrix(indi,false);
	rc = pthread_mutex_unlock(&mtx);

	if (ga_phase == 1) {
		maxcount = (int(pow(pow(myfinalG->valuedness, myfinalG->numIO),2)));
		if (hipCrealf(indi->phase) != 0 && hipCimagf(indi->phase) != 0) {
			for (int k = 0; k < maxcount; k++)
					myfinalG->gateMatrix1[k] = hipCmulf(
							myfinalG->gateMatrix1[k], indi->phase);
		}
	}

	//propagate each input through the circuit
	for (int k = 0; k < l; k++) {

		for (int i = 0; i < l; i++) {
			inter0[i] = make_hipFloatComplex(0, 0);
			inter1[i] = make_hipFloatComplex(0, 0);
		}

		//initialize variables
		for (int w = 0; w < l; w++)
			sta[w] = inter0[w] = inter1[w] = logicresult[w]
			                                             = make_hipFloatComplex(0, 0);
		//set the orthonormal state - k
		sta[k] = make_hipFloatComplex(1, 0);

		//propagate the state through the matrix
		for (int i = 0; i < l; i++) {
			for (int j = 0; j < l; j++) {
				logicresult[i] = hipCaddf(logicresult[i], hipCmulf(sta[j],
						myfinalG->gateMatrix1[i*l+j]));
			}
		}

		//init the ga_measurement operator with respect to the desired output
		//for each measured ouptut state get the expectation values
		expect0 = expectationsAllState[k][0];
		expect1 = expectationsAllState[k][1];

		//apply the ga_measurement operator for the desired state
		for (int i = 0; i < l; i++) {
			//rows
			for (int j = 0; j < l; j++) {
				inter0[i] = hipCaddf(inter0[i], hipCmulf(logicresult[j],
						measurementsAllState[2* k ]->gateMatrix1[i*l+j]));
				inter1[i] = hipCaddf(inter1[i], hipCmulf(logicresult[j],
						measurementsAllState[2* k + 1]->gateMatrix1[i*l+j]));
			}
		}
		//p(0) - not really p(0) but rather the desired result ---------------------------------
		expectation0 = make_hipFloatComplex(0, 0);
		ga_alphas = make_hipFloatComplex(0, 0);
		//the finnal inner product for p(0)
		for (int j = 0; j < l; j++)
			expectation0 = hipCaddf(expectation0, hipCmulf(
					hipConjf(logicresult[j]), inter0[j]));
		//state after the ga_measurement for 0
		expe_0
		= complex<float> (hipCrealf(expectation0),
				hipCimagf(expectation0));
		expe_0 = sqrt(expe_0);
		expectation0 = make_hipFloatComplex(real(expe_0), imag(expe_0));

		for (int i = 0; i < l; i++) {
			inter0[i] = hipCdivf(inter0[i], expectation0);
			ga_alphas = hipCaddf(ga_alphas, inter0[i]);
		}
		//p(1) ---------------------------------------------------------------------------------
		//vector inner product
		expectation1 = make_hipFloatComplex(0, 0);
		ga_betas = make_hipFloatComplex(0, 0);
		//the finnal inner product for p(1)
		for (int i = 0; i < l; i++)
			expectation1 = hipCaddf(expectation1, hipCmulf(
					hipConjf(logicresult[i]), inter1[i]));
		//state after the ga_measurement for 1
		expe_1
		= complex<float> (hipCrealf(expectation1),
				hipCimagf(expectation1));
		expe_1 = sqrt(expe_1);
		expectation0 = make_hipFloatComplex(real(expe_1), imag(expe_1));
		for (int i = 0; i < l; i++) {
			inter1[i] = hipCdivf(inter1[i], expectation1);
			ga_betas = hipCaddf(ga_betas, inter1[i]);
		}
		//--------------------------------------------------------------------------------------
		ga_alphas = expectation0;
		ga_betas = expectation1;

		//		cout<<"ga_alpha: "<<ga_alphas<<" + ga_beta: "<<ga_betas<<endl;
		//calculate the total state
		//Total State = M(0)+M(1)State/Measured(0)+Measured(1)
		mes = 0;
		resultingTotal[k][2* mes ] = expectation0;
		resultingTotal[k][2* mes + 1] = expectation1;
		if (hipCimagf(measureexpected[k][2* mes ]) == 1 && hipCimagf(
				measureexpected[k][2* mes + 1]) == 1) {
			//skip don't cares
			resultingTotalError[k][2* mes ] = make_hipFloatComplex(0, 0);
			resultingTotalError[k][2* mes + 1] = make_hipFloatComplex(0, 0);
		} else {
			if (hipCrealf(expectationsAllState[k][mes]) == hipCrealf(
					expectationsAllState[k][mes + 1])) {
				if (hipCrealf(expectationsAllState[k][mes]) == 1 || hipCrealf(
						expectationsAllState[k][mes]) == 0) {
					if ((hipCrealf(hipCaddf(ga_alphas, ga_betas)) == 1) && (hipCrealf(
							ga_alphas) == hipCrealf(ga_betas))) {
						resultingTotalError[k][mes * 2] = make_hipFloatComplex(
								0, 0);
						resultingTotalError[k][mes * 2 + 1]
						                       = make_hipFloatComplex(0, 0);
					} else {
						resultingTotalError[k][mes * 2] = make_hipFloatComplex(
								abs(0.5 - hipCabsf(expectation0)), 0);
						resultingTotalError[k][mes * 2 + 1]
						                       = make_hipFloatComplex(abs(0.5 - hipCabsf(
						                    		   expectation1)), 0);
					}
				} else {
					resultingTotalError[k][mes * 2] = make_hipFloatComplex(abs(
							hipCabsf(expect0) - hipCabsf(expectation0)), 0);
					resultingTotalError[k][mes * 2 + 1] = make_hipFloatComplex(
							abs(hipCabsf(expect1) - hipCabsf(expectation1)), 0);
				}
			} else {
				resultingTotalError[k][mes * 2] = make_hipFloatComplex(abs(
						hipCabsf(expect0) - hipCabsf(expectation0)), 0);
				resultingTotalError[k][mes * 2 + 1] = make_hipFloatComplex(abs(
						hipCabsf(expect1) - hipCabsf(expectation1)), 0);
			}

		}
	}

	int m = 0;
	inter = make_hipFloatComplex(0, 0);
	for (int e = 0; e < l; e++) {
		//		for (int c = 0; c < ga_measurement; c++){
		if (!(hipCimagf(measureexpected[0][e]) == 1 && hipCimagf(
				measureexpected[1][e]) == 1)) {
			//get the error over both possible ga_measurements of the state - for 0 and 1 of the first qubit
			inter = hipCaddf(inter, hipCdivf(hipCaddf(resultingTotalError[e][0],
					resultingTotalError[e][1]), make_hipFloatComplex(1, 0)));
			//expecting the desired higher probabilities 1
			m++;
		}
		//		}
	}
	//indi->Error /= ga_measurement;
	indi->Error = hipCrealf(hipCdivf(inter, make_hipFloatComplex(m, 0)));
	indi->Cost = (exp(-pow((ga_divider - indi->Cost), 2)));

	//generate fitness value
	if (indi->Error != 0) {
		switch (ga_replicator) {
		case 0:
			//simple fitness1
			indi->fitness = (1 - indi->Error);
			break;
		case 1:
			//simple fitness1
			indi->fitness = (1 / (indi->Error + 1));
			break;
		case 2:
			//scaled complex fitness1
			indi->fitness = (ga_alpha * (1 - indi->Error) + ga_beta * indi->Cost);
			break;
		case 3:
			//scaled complex fitness2
			indi->fitness = (ga_alpha * (1 / (indi->Error + 1)) + ga_beta
					* indi->Cost);
			break;
		case 4:
			indi->fitness = (1 - indi->Error);
			break;
		}
	} else {
		indi->Error = 0;
		indi->fitness = 1;//we found a good individual;
	}
	//	cout<<"Fitness: "<<indi->fitness<<endl;
	destroyGate(myfinalG);
	delete (myfinalG);
	return 0;
}
